#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
# define M 100000
# define N 100000
# define K 100000

__global__ void add(float *A, float *B, float *C){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    C[index] = A[index] + B[index];
}


int main(){
    int size = 9;
    float h_A[size] = {1., 1., 1., 1., 1., 1., 1., 1., 1.};
    float h_B[size] ={1., 1., 1., 1., 1., 1., 1., 1., 1.};
    float *A = &h_A[0];
    float *B = &h_B[0];
    float *h_C = (float*)malloc(sizeof(float)* size);
    float *d_A, *d_B, *d_C;
    
    hipMalloc((void **)&d_C, size * sizeof(float));
    hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    add<<<1 , size>>>(d_A, d_B, d_C);
    hipMemcpy(h_C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < size ; i++ ){
        printf("%f", h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}