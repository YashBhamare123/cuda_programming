#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
# define M 100000
# define N 100000
# define K 100000

__global__ void multiply(float *A, float *B, float *C, int m, int n, int k){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int offset_A = index / k;
    int offset_B = index % k;
    if (index < m * n){
        C[index] = 0;
        for (int i = 0; i < k; i++){
            C[index] += A[offset_A * k + i]*B[offset_B + i * k];
        }
    }
}

int main(){
    int num = 9;
    size_t size = num * sizeof(float);
    float h_A[num] = {1., 1., 1., 1., 1., 1., 1., 1., 1.};
    float h_B[num] ={1., 1., 1., 1., 1., 1., 1., 1., 1.};
    float *h_C = (float*)malloc(size);
    float *d_A, *d_B, *d_C;
    

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


    multiply<<<1 ,num>>>(d_A, d_B, d_C, 3, 3, 3);

    hipDeviceSynchronize(); // redundant cause memcpy does implicit device sync
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < num; i++ ){
        printf("%f\n", h_C[i]);
    }


    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


