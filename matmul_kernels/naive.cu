#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
# define M 100000
# define N 100000
# define K 100000

__global__ void multiply(float *A, float *B, float *C, int m, int n, int k){
    int index = blockId.x * blockDim + threadId.x;
    int offset_A = index / k;
    int offset_B = index;
    if (index < m * n){
        C[index] = 0;
        for (int i = 0; i < k, i++){
            C[index] += A[offset_A + i]*B[offset_B + i * k];
        }
    }
}

int main(){
    int size = 9;
    float h_A[size] = {1., 1., 1., 1., 1., 1., 1., 1., 1.};
    float h_B[size] ={1., 1., 1., 1., 1., 1., 1., 1., 1.};
    float *h_C = (float*)malloc(sizeof(float)* size);
    float *d_A, *d_B, *d_C;
    
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_A, h_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, hipMemcpyHostToDevice);

    multiply<<<1 , size>>(d_A, d_B, d_C, 3, 3, 3);

    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, hipMemcpyDeviceToHost);
    for (int i = 0; i < size, i++ ){
        cout<<h[i]<<endl;
    }


    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


